#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include<algorithm>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include<iostream>
#include "CycleTimer.h"

using namespace std;
extern float toBW(int bytes, float sec);


//帮助函数四舍五入到2的幂
 
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void exclusive_scan_parallel1(int* device_data, int twod, int N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    index *= twod;

    if(index < N)
        device_data[index + twod-1] += device_data[index + twod/2 - 1];

}

__global__ void exclusive_scan_parallel2(int* device_data, int twod, int N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index == 0 && twod == N){
        device_data[N-1] = 0;
    }
    __syncthreads();
    index *= twod;
    if(index < N){
        int t = device_data[index + twod/2 - 1];
        device_data[index + twod/2 - 1] = device_data[index + twod - 1];
        device_data[index + twod - 1] += t;
    }
}

void exclusive_scan(int* device_data, int length)
{

    const int threadsPerBlock = 512;
    int N = length;
    for (int twod = 1; twod < N/2; twod*=2)
    {
        int twod1 = twod*2;
        exclusive_scan_parallel1<<<(N/twod1 + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock>>>(device_data, twod1, N);
        hipDeviceSynchronize();
        // parallel_for (int i = 0; i < N; i += twod1)
        // data[i+twod1-1] += data[i+twod-1];
    }
    // // downsweep phase.
    for (int twod = N/2; twod >= 1; twod /= 2)
    {
        int twod1 = twod*2;
        exclusive_scan_parallel2<<<(N/twod1 + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock>>>(device_data, twod1, N);
        hipDeviceSynchronize();
    }


}
// void exclusive_scan_iterative(int* device_data, int length)
// {
//     int N = length;
//     // upsweep phase.
//     for (int twod = 1; twod < N; twod*=2)
//     {
//         int twod1 = twod*2;
//         exclusive_scan_parallel1<<<1, N/twod1>>>(device_data, twod1, N);
//         hipDeviceSynchronize();
//         // parallel_for (int i = 0; i < N; i += twod1)
//         // data[i+twod1-1] += data[i+twod-1];
//     }
//     device_data[N-1] = 0;
//     // downsweep phase.
//     for (int twod = N/2; twod >= 1; twod /= 2)
//     {
//         int twod1 = twod*2;
//         exclusive_scan_parallel2<<<1, N/twod1>>>(device_data, twod1, N);
//         hipDeviceSynchronize();
//         // parallel_for (int i = 0; i < N; i += twod1)
//         // {
//         //     int t = data[i+twod-1];
//         //     data[i+twod-1] = data[i+twod1-1];
//         //     data[i+twod1-1] += t;
//         // }
//     }
// }

/*  这个函数是你将要编写的代码的包装器——它将输入复制到 GPU，并且 exclusive_scan()函数的调用次数。你不应该修改它。
 */
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_data;
    //我们将数组的大小四舍五入到2的幂，但元素在2之后
    //初始输入的末尾未初始化，也不检查是否正确。
    //如果你假设数组的长度是2的幂，但这将导致在非2的幂输入上增加额外的工作。
    int rounded_length = nextPow2(end - inarray);
    hipMalloc((void **)&device_data, sizeof(int) * rounded_length);

    hipMemcpy(device_data, inarray, (end - inarray) * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_data, rounded_length);

    // Wait for any work left over to be completed.
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    
    hipMemcpy(resultarray, device_data, (end - inarray) * sizeof(int),
               hipMemcpyDeviceToHost);
    return overallDuration;
}


// 包装 Thrust 库独有的扫描函数 如上所述，将输入复制到 GPU 上，并且只计算扫描本身的执行时间。预计您不会产生与 Thrust 版本相比具有竞争力的性能。
 
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), 
               hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int),
               hipMemcpyDeviceToHost);
    thrust::device_free(d_input);
    thrust::device_free(d_output);
    double overallDuration = endTime - startTime;
    return overallDuration;
}

__global__ void find_cuda_peaks(int *device_input, int N, int *device_output, int* device_output_length){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N - 1 && index > 0){
        if(device_input[index - 1] < device_input[index] && device_input[index] > device_input[index + 1]){
            device_output[index] = index;
            atomicAdd(device_output_length,1);
        }else{
            device_output[index] = 999999;
        }
    }else{
        device_output[index] = 999999;
    }
    

}


void find_peaks(int *device_input, int length, int *device_output, int *device_output_length) {
/* TODO：
     * 查找列表中大于前后元素的所有元素，
     * 将元素的索引存储到 device_result 中。
     * 返回找到的峰元素数。
     * 根据定义，元素 0 和元素 length-1 都不是峰值。
     *
     * 你的任务是实现这个功能。你可能想要
     * 利用一个或多个对 exclusive_scan() 的调用，以及
     * 额外的 CUDA 内核启动。
     * 注意：与扫描代码一样，我们确保分配的数组是一个幂
     * 大小为 2，因此您可以使用您的 Exclusive_scan 功能，如果
     * 它需要。但是，您必须确保结果
     * 给定原始长度，find_peaks 是正确的。
     */
    int threadsPerBlock = 64;

    find_cuda_peaks<<<(length + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock>>>(device_input,length,device_output,device_output_length);
    
}



/* Timing wrapper around find_peaks. You should not modify this function.
 */
double cudaFindPeaks(int *input, int length, int *output, int *output_length) {
    int *device_input;
    int *device_output;
    int *device_output_length;
    int rounded_length = length;
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output_length, 1 * sizeof(int));
    hipMemcpy(device_output_length, output_length, 1 * sizeof(int), 
               hipMemcpyHostToDevice);    
    hipMemcpy(device_input, input, length * sizeof(int), 
               hipMemcpyHostToDevice);


    double startTime = CycleTimer::currentSeconds();
    
    find_peaks(device_input, rounded_length, device_output,device_output_length);
    

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(output_length, device_output_length, 1 * sizeof(int),
               hipMemcpyDeviceToHost);

    hipMemcpy(output, device_output, rounded_length * sizeof(int),
               hipMemcpyDeviceToHost);
    sort(output,output+rounded_length);
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_output_length);

    return endTime - startTime;
}


void printCudaInfo()
{
    // 为了好玩，只打印出机器上的一些统计数据

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
