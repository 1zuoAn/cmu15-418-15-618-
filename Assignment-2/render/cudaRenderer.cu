#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>



#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"


////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// This stores the global constants
struct GlobalConstants {

    SceneName sceneName;

    int numberOfCircles;

    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;
};

//所有 cuda 内核的范围内但只读的全局变量。常量修饰符指定这个变量将存储在 GPU 上的特殊“常量”内存中。
//(我们在课堂上没有讨论这种类型的内存，但是常量内存是放置只读变量的一个快捷的地方)。
__constant__ GlobalConstants cuConstRendererParams;

// 只读查找表用于快速计算噪声(雪花场景的 Advanced Animation 需要)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// 颜色斜坡查找着色器所需的颜色斜坡表
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// 包括来自外部文件的 CUDA 代码的一部分，以保持这个文件更简单，并分离不应该被修改的代码
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"
#include "circleBoxTest.cu_inl"




// kernelClearImageSnowflake --  (CUDA 设备代码)
//
// 清除图像，将图像设置为雪花图像中使用的白灰渐变
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // 写入全局内存: 作为一种优化，这段代码使用了 float4存储，这比将它编码为四个独立的 float 存储更有效率。
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// 清除图像，将所有像素设置为指定的颜色 rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // 写入全局内存: 作为一种优化，这段代码使用 float4//store，这比将它编码为//四个单独的 float store 更有效率
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// 内核
// 
// 更新烟花的位置
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = M_PI;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update 
        return;
    }

    // 测定烟花中心/火花指数
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // Update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // Firework sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // 利用烟花火花计算矢量
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // 计算与烟火的距离
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // 恢复到起始位置
        //  烟花边缘随机起动位置
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // 行程缩短单位长度
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis   
//
// 更新圆圈的半径/颜色
__global__ void kernelAdvanceHypnosis() { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles) 
        return; 

    float* radius = cuConstRendererParams.radius; 

    float cutOff = 0.5f;
    // 到达阈值半径后将圆放回中心
    if (radius[index] > cutOff) { 
        radius[index] = 0.02f; 
    } else { 
        radius[index] += 0.01f; 
    }   
}   


// kernelAdvanceBouncingBalls
// 
// 更新球的位置
__global__ void kernelAdvanceBouncingBalls() { 
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x; 
   
    if (index >= cuConstRendererParams.numberOfCircles) 
        return; 

    float* velocity = cuConstRendererParams.velocity; 
    float* position = cuConstRendererParams.position; 

    int index3 = 3 * index;
    // 如果中心位置 < 0，则反向速度
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // 停止状态
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // 弹球
        velocity[index3+1] *= kDragCoeff;
    }

    // 更新速度: v = u + at (仅沿 y 轴)
    velocity[index3+1] += kGravity * dt;

    // 更新位置(仅沿 y 轴)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball 
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// 将雪花动画向前移动一个时间步骤。更新圆周位置和速度。注意当雪花从屏幕的左边、右边或底部移动时，它的位置是如何重置的。
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // 从全局内存加载
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // 黑客使更远的圆移动得更慢，给人视差错觉
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // 在动作中加入一些噪音，使雪飘动
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // Drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // Update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // Update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // 如果雪花已经从屏幕的左边、右边或底部移动，把它放回顶部，给它随机的 x 位置和速度。
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // 从0垂直速度重新开始。选择一个伪随机水平速度。
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // 将更新的位置和速度存储到全局内存中
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// 给定一个像素和一个圆，确定圆对像素的贡献。图像的更新是在这个函数中完成的。由 kernelRenderCircles ()调用
__device__ __inline__ void
shadePixel(float2 pixelCenter, float3 p, float4* imagePtr, int circleIndex,int* bLock, int* mLock) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];
    float maxDist = rad * rad;

    // 圆圈对图像没有贡献
    if (pixelDist > maxDist)
        return;


    float3 rgb;
    float alpha;

    // 有一个非零贡献。现在计算阴影值

    // 建议: 这个条件在内部循环中。尽管它对所有线程的计算结果都是一样的，但是为了实现条件，需要设置行道掩码等等。
    //明智的做法是在 inkernelRenderCircles 循环之外执行此逻辑。(如果你自我感觉良好，你可以使用一些专门的模板魔术)。
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // Simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;
    bool blocked = true;
    // 开始应该是原子的区域
    // 全局内存读取

    for(int i = circleIndex-1;i >= 0; i--){
        //判断是否像素点对优先级更大的圆有贡献
        float3 p = *(float3*)(&cuConstRendererParams.position[3*i]);
        if(circleInBoxConservative(p.x,p.y,cuConstRendererParams.radius[i],
        pixelCenter.x,pixelCenter.x,pixelCenter.y,pixelCenter.y)){
            blocked = true;
            while(blocked){
                //询问大锁，是否上锁了
                if(1 == atomicCAS(mLock+i,0,0)){
                    blocked = false;
                }
            }
        }
    }
    blocked = true;
    while(blocked){
       // printf("wait\n");
        if(1 == atomicCAS(bLock,0,1)){
        // printf("pass\n");
            float4 existingColor = *imagePtr;
            float4 newColor;
            newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
            newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
            newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
            newColor.w = alpha + existingColor.w;

            // 全局内存写入
            *imagePtr = newColor;
            atomicExch(bLock,0);
            blocked = false;
        }
        
    }

    //结束应该是原子的区域

}

// kernelRenderCircles -- (CUDA device code)
//
//每个线程呈现一个圆。由于没有任何保护措施来确保输出图像的更新或互斥锁顺序，因此产生的图像将是不正确的。
__global__ void kernelRenderCircles(int* bLock,int* mLock) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;


    if (index >= cuConstRendererParams.numberOfCircles)
        return;


    atomicExch(mLock+index,1);
    int index3 = 3 * index;

    // 读取位置和半径
    float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
    float  rad = cuConstRendererParams.radius[index];

    // 计算圆的边框。边界是整数屏幕坐标，所以它被夹在屏幕的边缘。
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // A bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    // 对于边界框中的所有像素
    for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]);
        for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
            float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
            shadePixel(pixelCenterNorm, p, imgPtr, index, bLock, mLock);
            imgPtr++;
        }
    }
    atomicExch(mLock+index,0);
}

////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numberOfCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image*
CudaRenderer::getImage() {

    // Need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numberOfCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce RTX 2080") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numberOfCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numberOfCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numberOfCircles = numberOfCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // Also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // Copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numberOfCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) { 
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>(); 
    }
    hipDeviceSynchronize();
}

void
CudaRenderer::render() {
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numberOfCircles + blockDim.x - 1) / blockDim.x);

    int *mLock = new int[numberOfCircles];
    memset(mLock,0,sizeof(mLock));
    hipMalloc((void **)&mLock, numberOfCircles * sizeof(int));
    int *bLock = new int[1];
        printf("pass\n\n");
    memset(bLock,0,sizeof(bLock));
    hipMalloc((void **)&bLock, 1 * sizeof(int));
    kernelRenderCircles<<<gridDim, blockDim>>>(bLock,mLock);
    hipDeviceSynchronize();
    hipFree(mLock);
    hipFree(bLock);
}
